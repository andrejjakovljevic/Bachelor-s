#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "BasicOperations.h"
using namespace std;

// Number of threads in each thread block
static const int blockSize = 1024;
static const int TILE_DIM = 32;
//static const long long max_size = (long long)3*1024*1024*1024;
int trensize = 0;


__global__ void vecAddDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecMulDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecSubDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntDoubleGPU(int *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}


__global__ void vecAddDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecAddIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecSubDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecMulDoubleGPU(double* a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecMulIntGPU(int* a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

void vecAddDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecAddIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulIntCPU(int *a, int *b, int *c, int n)
{
     // int *d_a;
      //int *d_b;
      //int *d_c;
      //int bytes = sizeof(int)*n;
      //float time;
      //hipMalloc(&d_a, bytes);
      //hipMalloc(&d_b, bytes);
      //hipMalloc(&d_c, bytes);
      //hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      //hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);

      /*hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop); 
      hipEventRecord(start, 0);
*/
      vecMulIntGPU<<<gridSize, blockSize>>>(a, b, c, n);

      //hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      /*hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);*/
      //hipFree(d_a);
      //hipFree(d_b);
      //hipFree(d_c);
      //printf("Time to generate:  %3.1f ms \n", time);
}

void vecAddDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntDoubleCPU(int *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void initVecIntCPU(int *a, int x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

void initVecDoubleCPU(double* a, double x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}



__global__ void dotIntGPU(int* A, int* B, int* C, int ARows, int ACols, int BRows,
      int BCols, int CRows, int CCols)
{
      int CValue = 0;

      int Row = blockIdx.y*TILE_DIM + threadIdx.y;
      int Col = blockIdx.x*TILE_DIM + threadIdx.x;

      __shared__ int As[TILE_DIM][TILE_DIM];
      __shared__ int Bs[TILE_DIM][TILE_DIM];

      for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

            if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
                  As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
            else
                  As[threadIdx.y][threadIdx.x] = 0.0;

            if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
            {
                  Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
            }
            else
                  Bs[threadIdx.y][threadIdx.x] = 0.0;

            __syncthreads();

            for (int n = 0; n < TILE_DIM; ++n)
                  CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

            __syncthreads();
      }

      if (Row < CRows && Col < CCols)
      {
            C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
                  (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
      }
}
  

__global__ void sumCommMultiBlockInt(const int *gArr, int n, int *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      int sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ int shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

__global__ void sumCommMultiBlockDouble(const double *gArr, int n, double *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      double sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ double shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) 
      { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

int vecSumInt(int* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      int out;
      int* dev_out;
      hipMalloc(&dev_out, sizeof(int)*gridSize);

      sumCommMultiBlockInt<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockInt<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

double vecSumDouble(double* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      double out;
      double* dev_out;
      hipMalloc(&dev_out, sizeof(double)*gridSize);

      sumCommMultiBlockDouble<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockDouble<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

bool check_error()
{
      hipError_t err = hipGetLastError();  
      return ( err != hipSuccess );
}

void copy_back(void* data, void*& d_data, int size)
{
      //hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
      hipFree(d_data);
      trensize-=size;
}

void copy_front(void* data, void*& d_data, int size)
{
      trensize+=size;
      hipMalloc(&d_data, size);
      //hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void g_alloc(void*& d_data, int size)
{
      hipMalloc(&d_data, size);
}

void just_return(void* data, void*& d_data, int size)
{
      hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
}

void just_front(void* data, void*& d_data, int size)
{
      hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void dot_prodIntCPU(int* a, int* b, int* c, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)BCols/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)ARows/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      dotIntGPU << < Grid_dim, Block_dim >> > (a, b, c, ARows, ACols, BRows, BCols, CRows, CCols);
}

__global__ void spliceIntGPU(int* a, int* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
      {
            a[id]=b[id];
      }
}

__global__ void spliceDoubleGPU(double* a, double* b, int n )
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            a[id]=b[id];
}

void* spliceInt(int* arr1, int start, int stop)
{
      void* d;
      int n = stop-start;
      int size = sizeof(int)*(stop-start);
      hipMalloc(&d, size);
      int gridSize = (int)ceil((float)n/blockSize);
      spliceIntGPU<<<gridSize, blockSize>>>((int*)d, arr1+start, n);
      return d;
}

void* spliceDouble(double* arr1, int start, int stop)
{
      void* d;
      int n = stop-start;
      int size = sizeof(double)*(stop-start);
      hipMalloc(&d, size);
      int gridSize = (int)ceil((float)n/blockSize);
      spliceDoubleGPU<<<gridSize, blockSize>>>((double*)d, arr1+start, n);
      return d;
}

void rangeSetInt(int* arr1, int* arr2, int start, int stop)
{
      int n = stop-start;
      int gridSize = (int)ceil((float)n/blockSize);
      spliceIntGPU<<<gridSize, blockSize>>>(arr1+start, arr2, n);
}

void rangeSetDouble(double* arr1, double* arr2, int start, int stop)
{
      int n = stop-start;
      int gridSize = (int)ceil((float)n/blockSize);
      spliceDoubleGPU<<<gridSize, blockSize>>>(arr1, arr2, n);
}