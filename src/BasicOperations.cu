#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "BasicOperations.h"
using namespace std;

// Number of threads in each thread block
static const int blockSize = 1024;
 
__global__ void vecAddDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + (double)b[id];
}

__global__ void vecMulDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecSubDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntDoubleGPU(int *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}


__global__ void vecAddDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecAddIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecSubDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecMulDoubleGPU(double* a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecMulIntGPU(int* a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

void vecAddDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);

      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecAddIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecAddDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubIntDoubleCPU(int *a, double *b, double *c, int n)
{
      int *d_a;
      double *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes1);
      hipMalloc(&d_b, bytes2);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes1, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes2, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void initVecIntCPU(int *a, int x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

void initVecDoubleCPU(double* a, double x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

__global__ void sumCommMultiBlockInt(const int *gArr, int n, int *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      int sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ int shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

__global__ void sumCommMultiBlockDouble(const double *gArr, int n, double *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      double sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ double shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

int vecSumInt(int* arr, int n) 
{
      int* dev_arr;
      int gridSize = (int)ceil((float)n/blockSize);
      hipMalloc(&dev_arr, n * sizeof(int));
      hipMemcpy(dev_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

      int out;
      int* dev_out;
      hipMalloc(&dev_out, sizeof(int)*gridSize);

      sumCommMultiBlockInt<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockInt<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
      hipFree(dev_arr);
      hipFree(dev_out);
      return out;
}

double vecSumDouble(double* arr, int n) 
{
      double* dev_arr;
      int gridSize = (int)ceil((float)n/blockSize);
      hipMalloc(&dev_arr, n * sizeof(double));
      hipMemcpy(dev_arr, arr, n * sizeof(double), hipMemcpyHostToDevice);

      double out;
      double* dev_out;
      hipMalloc(&dev_out, sizeof(double)*gridSize);

      sumCommMultiBlockDouble<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockDouble<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost);
      hipFree(dev_arr);
      hipFree(dev_out);
      return out;
}