#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "BasicOperations.h"
using namespace std;

// Number of threads in each thread block
static const int blockSize = 1024;
static const int TILE_DIM = 32;
//static const long long max_size = (long long)3*1024*1024*1024;
int trensize = 0;


__global__ void vecAddDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecMulDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecSubDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntDoubleGPU(int *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}


__global__ void vecAddDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecAddIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecSubDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecMulDoubleGPU(double* a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecMulIntGPU(int* a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecDivDoubleGPU(double* a, double* b, double* c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] / b[id];
}

__global__ void vecDivIntGPU(int* a, int* b, double* c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = (double)a[id] / (double)b[id];
}

__global__ void vecDivDoubleIntGPU(double* a, int* b, double* c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] / (double)b[id];
}

__global__ void vecDivIntDoubleGPU(int* a, double* b, double* c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = (double)a[id] / b[id];
}

void vecDivDoubleCPU(double* a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecDivDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);      
}

void vecDivIntCPU(int* a, int* b, double* c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecDivIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecDivDoubleIntCPU(double* a, int* b, double* c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecDivDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecDivIntDoubleCPU(int* a, double* b, double* c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecDivIntDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecAddDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecAddIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulIntCPU(int *a, int *b, int *c, int n)
{
     // int *d_a;
      //int *d_b;
      //int *d_c;
      //int bytes = sizeof(int)*n;
      //float time;
      //hipMalloc(&d_a, bytes);
      //hipMalloc(&d_b, bytes);
      //hipMalloc(&d_c, bytes);
      //hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      //hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);

      /*hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop); 
      hipEventRecord(start, 0);
*/
      vecMulIntGPU<<<gridSize, blockSize>>>(a, b, c, n);

      //hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      /*hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);*/
      //hipFree(d_a);
      //hipFree(d_b);
      //hipFree(d_c);
      //printf("Time to generate:  %3.1f ms \n", time);
}

void vecAddDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntDoubleCPU(int *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void initVecIntCPU(int *a, int x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

void initVecDoubleCPU(double* a, double x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}



__global__ void dotIntGPU(int* A, int* B, int* C, int ARows, int ACols, int BRows,
      int BCols, int CRows, int CCols)
{
      int CValue = 0;

      int Row = blockIdx.y*TILE_DIM + threadIdx.y;
      int Col = blockIdx.x*TILE_DIM + threadIdx.x;

      __shared__ int As[TILE_DIM][TILE_DIM];
      __shared__ int Bs[TILE_DIM][TILE_DIM];

      for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

            if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
                  As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
            else
                  As[threadIdx.y][threadIdx.x] = 0.0;

            if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
            {
                  Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
            }
            else
                  Bs[threadIdx.y][threadIdx.x] = 0.0;

            __syncthreads();

            for (int n = 0; n < TILE_DIM; ++n)
                  CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

            __syncthreads();
      }

      if (Row < CRows && Col < CCols)
      {
            C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
                  (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
      }
}

__global__ void dotDoubleGPU(double* A, double* B, double* C, int ARows, int ACols, int BRows,
      int BCols, int CRows, int CCols)
      {
            double CValue = 0;

            int Row = blockIdx.y*TILE_DIM + threadIdx.y;
            int Col = blockIdx.x*TILE_DIM + threadIdx.x;
      
            __shared__ double As[TILE_DIM][TILE_DIM];
            __shared__ double Bs[TILE_DIM][TILE_DIM];
      
            for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {
      
                  if (k*TILE_DIM + threadIdx.x < ACols && Row < ARows)
                        As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_DIM + threadIdx.x];
                  else
                        As[threadIdx.y][threadIdx.x] = 0.0;
      
                  if (k*TILE_DIM + threadIdx.y < BRows && Col < BCols)
                  {
                        Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*BCols + Col];
                  }
                  else
                        Bs[threadIdx.y][threadIdx.x] = 0.0;
      
                  __syncthreads();
      
                  for (int n = 0; n < TILE_DIM; ++n)
                        CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
      
                  __syncthreads();
            }
      
            if (Row < CRows && Col < CCols)
            {
                  C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
                        (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
            }
      }
  

__global__ void sumCommMultiBlockInt(const int *gArr, int n, int *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      int sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ int shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

__global__ void sumCommMultiBlockDouble(const double *gArr, int n, double *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      double sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ double shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) 
      { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

int vecSumInt(int* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      int out;
      int* dev_out;
      hipMalloc(&dev_out, sizeof(int)*gridSize);

      sumCommMultiBlockInt<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockInt<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

double vecSumDouble(double* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      double out;
      double* dev_out;
      hipMalloc(&dev_out, sizeof(double)*gridSize);

      sumCommMultiBlockDouble<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockDouble<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

bool check_error()
{
      hipError_t err = hipGetLastError();  
      //printf("CUDA Error: %s\n", hipGetErrorString(err));  
      return ( err != hipSuccess );
}

void copy_back(void* data, void*& d_data, int size)
{
      //hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
      hipFree(d_data);
      trensize-=size;
}

void copy_front(void* data, void*& d_data, int size)
{
      trensize+=size;
      hipMalloc(&d_data, size);
      //hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void g_alloc(void*& d_data, int size)
{
      hipMalloc(&d_data, size);
}

void just_return(void* data, void*& d_data, int size)
{
      hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
}

void just_front(void* data, void*& d_data, int size)
{
      hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void dot_prodIntCPU(int* a, int* b, int* c, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)BCols/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)ARows/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      dotIntGPU << < Grid_dim, Block_dim >> > (a, b, c, ARows, ACols, BRows, BCols, CRows, CCols);
}

void dot_prodDoubleCPU(double* a, double* b, double* c, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)BCols/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)ARows/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      dotDoubleGPU << < Grid_dim, Block_dim >> > (a, b, c, ARows, ACols, BRows, BCols, CRows, CCols);
}

__global__ void spliceIntGPU(int* a, int* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
      {
            a[id]=b[id];
      }
}

__global__ void spliceDoubleGPU(double* a, double* b, int n )
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            a[id]=b[id];
}

void* spliceInt(int* arr1, int start, int stop)
{
      void* d;
      int n = stop-start;
      int size = sizeof(int)*(stop-start);
      hipMalloc(&d, size);
      int gridSize = (int)ceil((float)n/blockSize);
      spliceIntGPU<<<gridSize, blockSize>>>((int*)d, arr1+start, n);
      return d;
}

void* spliceDouble(double* arr1, int start, int stop)
{
      void* d;
      int n = stop-start;
      int size = sizeof(double)*(stop-start);
      hipMalloc(&d, size);
      int gridSize = (int)ceil((float)n/blockSize);
      spliceDoubleGPU<<<gridSize, blockSize>>>((double*)d, arr1+start, n);
      return d;
}

void rangeSetInt(int* arr1, int* arr2, int start, int stop)
{
      int n = stop-start;
      int gridSize = (int)ceil((float)n/blockSize);
      spliceIntGPU<<<gridSize, blockSize>>>(arr1+start, arr2, n);
}

void rangeSetDouble(double* arr1, double* arr2, int start, int stop)
{
      int n = stop-start;
      int gridSize = (int)ceil((float)n/blockSize);
      spliceDoubleGPU<<<gridSize, blockSize>>>(arr1+start, arr2, n);
}

__global__ void transposeInplaceIntGPU(int* srcDst, int width, int pitch)
{
      int col = blockIdx.x * blockDim.x + threadIdx.x;
      int row = blockIdx.y * blockDim.y + threadIdx.y;

      int tid_in = row * pitch + col;
      int tid_out = col * pitch + row;

      if((row < width) && (col < width) && (row<col)) 
      {
            int temp = srcDst[tid_out];
            srcDst[tid_out] = srcDst[tid_in];
            srcDst[tid_in] = temp;
      }
} 

__global__ void transposeInplaceDoubleGPU(double* srcDst, int width, int pitch)
{
      int col = blockIdx.x * blockDim.x + threadIdx.x;
      int row = blockIdx.y * blockDim.y + threadIdx.y;

      int tid_in = row * pitch + col;
      int tid_out = col * pitch + row;

      if((row < width) && (col < width) && (row<col)) 
      {
            double temp = srcDst[tid_out];
            srcDst[tid_out] = srcDst[tid_in];
            srcDst[tid_in] = temp;
      }
} 

void transposeInplaceIntCPU(int* srcDst, int width, int height)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)width/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)height/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      transposeInplaceIntGPU << < Grid_dim, Block_dim >> > (srcDst, width, height);
}

void transposeInplaceDoubleCPU(double* srcDst, int width, int height)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)width/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)height/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      transposeInplaceDoubleGPU << < Grid_dim, Block_dim >> > (srcDst, width, height);
}

__global__ void mulScalarIntDoubleGPU(int* a, int x, double* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            b[id] = (double)a[id]*(double)x;
}

__global__ void divScalarIntDoubleGPU(int* a, int x, double* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            b[id] = (double)a[id]/(double)x;
}

__global__ void mulScalarDoubleDoubleGPU(double* a, double x, double* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            b[id] = (double)a[id]*(double)x;
}

__global__ void divScalarDoubleDoubleGPU(double* a, double x, double* b, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            b[id] = (double)a[id]/(double)x;
}

void mulScalarIntDoubleCPU(int* a, int x, double* b, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      mulScalarIntDoubleGPU<<<gridSize, blockSize>>>(a, x, b, n);
}

void divScalarIntDoubleCPU(int* a, int x, double* b, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      divScalarIntDoubleGPU<<<gridSize, blockSize>>>(a, x, b, n);
}


void mulScalarDoubleDoubleCPU(double* a, double x, double* b, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      mulScalarDoubleDoubleGPU<<<gridSize, blockSize>>>(a, x, b, n);
}

void divScalarDoubleDoubleCPU(double* a, double x, double* b, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      divScalarDoubleDoubleGPU<<<gridSize, blockSize>>>(a, x, b, n);
}

__global__ void getSubMatrixDoubleGPU(double* a, double* b, int x1, int y1, int xd, int yd, int x, int y)
{
      int y_moj = blockIdx.y*TILE_DIM + threadIdx.y;
      int x_moj = blockIdx.x*TILE_DIM + threadIdx.x;
      if (y_moj<yd && x_moj<xd)
      {
            b[y_moj*xd+x_moj]=a[(y_moj+y1)*x+x_moj+x1];
      }
}

__global__ void getSubMatrixIntGPU(int* a, int* b, int x1, int y1, int xd, int yd, int x, int y)
{
      int y_moj = blockIdx.y*TILE_DIM + threadIdx.y;
      int x_moj = blockIdx.x*TILE_DIM + threadIdx.x;
      if (y_moj<yd && x_moj<xd)
      {
            b[y_moj*xd+x_moj]=a[(y_moj+y1)*x+x_moj+x1];
      }
}

void getSubMatrixDoubleCPU(double* a, double* b, int x1, int y1, int xd, int yd, int x, int y)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)xd/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)yd/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      getSubMatrixDoubleGPU << < Grid_dim, Block_dim >> > (a, b, x1, y1, xd, yd, x, y);
}

void getSubMatrixIntCPU(int* a, int* b, int x1, int y1, int xd, int yd, int x, int y)
{
      dim3 Block_dim(TILE_DIM, TILE_DIM);
      int gridSizeWidth1 = (int)ceil((float)xd/TILE_DIM);
      int gridSizeWidth2 = (int)ceil((float)yd/TILE_DIM);
      dim3 Grid_dim(gridSizeWidth1, gridSizeWidth2);
      getSubMatrixIntGPU << < Grid_dim, Block_dim >> > (a, b, x1, y1, xd, yd, x, y);
}