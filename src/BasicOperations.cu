#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include "BasicOperations.h"
using namespace std;

// Number of threads in each thread block
static const int blockSize = 1024;
static const int BLOCK_SIZE = 32;
//static const long long max_size = (long long)3*1024*1024*1024;
int trensize = 0;


__global__ void vecAddDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecMulDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecSubDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntDoubleGPU(int *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}


__global__ void vecAddDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecAddIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecSubDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecMulDoubleGPU(double* a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecMulIntGPU(int* a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

void vecAddDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleCPU(double *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecAddIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntCPU(int *a, int *b, int *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulIntCPU(int *a, int *b, int *c, int n)
{
     // int *d_a;
      //int *d_b;
      //int *d_c;
      //int bytes = sizeof(int)*n;
      //float time;
      //hipMalloc(&d_a, bytes);
      //hipMalloc(&d_b, bytes);
      //hipMalloc(&d_c, bytes);
      //hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      //hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);

      /*hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop); 
      hipEventRecord(start, 0);
*/
      vecMulIntGPU<<<gridSize, blockSize>>>(a, b, c, n);

      //hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      /*hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time, start, stop);*/
      //hipFree(d_a);
      //hipFree(d_b);
      //hipFree(d_c);
      //printf("Time to generate:  %3.1f ms \n", time);
}

void vecAddDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecMulDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubDoubleIntCPU(double *a, int *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleIntGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void vecSubIntDoubleCPU(int *a, double *b, double *c, int n)
{
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntDoubleGPU<<<gridSize, blockSize>>>(a, b, c, n);
}

void initVecIntCPU(int *a, int x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

void initVecDoubleCPU(double* a, double x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

__global__ void dotIntGPU(int* d_a, int* d_b, int* d_c, int n) 
{
      __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
      __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

      int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
      int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
      int tmp = 0;
      int idx;

      for (int sub = 0; sub < gridDim.x; ++sub) 
      {
            idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
            if(idx >= n*n)
            {
                  tile_a[threadIdx.y][threadIdx.x] = 0;
            }
            else
            {
                  tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
            }

            idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
            if(idx >= n*n)
            {
                  tile_b[threadIdx.y][threadIdx.x] = 0;
            }  
            else
            {
                  tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
            }
            __syncthreads();

            for (int k = 0; k < BLOCK_SIZE; ++k) 
            {
                  tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
            }
            __syncthreads();
      }
      if(row < n && col < n)
      {
            d_c[row * n + col] = tmp;
      }
}

__global__ void sumCommMultiBlockInt(const int *gArr, int n, int *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      int sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ int shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

__global__ void sumCommMultiBlockDouble(const double *gArr, int n, double *gOut) 
{
      int thIdx = threadIdx.x;
      int gthIdx = thIdx + blockIdx.x*blockSize;
      const int gridSize = blockSize*gridDim.x;
      double sum = 0;
      for (int i = gthIdx; i < n; i += gridSize)
            sum += gArr[i];
      __shared__ double shArr[blockSize];
      shArr[thIdx] = sum;
      __syncthreads();
      for (int size = blockSize/2; size>0; size/=2) 
      { //uniform
            if (thIdx<size)
                  shArr[thIdx] += shArr[thIdx+size];
            __syncthreads();
      }
      if (thIdx == 0)
            gOut[blockIdx.x] = shArr[0];
}

int vecSumInt(int* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      int out;
      int* dev_out;
      hipMalloc(&dev_out, sizeof(int)*gridSize);

      sumCommMultiBlockInt<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockInt<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

double vecSumDouble(double* dev_arr, int n) 
{
      int gridSize = (int)ceil((float)n/blockSize);

      double out;
      double* dev_out;
      hipMalloc(&dev_out, sizeof(double)*gridSize);

      sumCommMultiBlockDouble<<<gridSize, blockSize>>>(dev_arr, n, dev_out);
      sumCommMultiBlockDouble<<<1, blockSize>>>(dev_out, gridSize, dev_out);
      hipDeviceSynchronize();

      hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost);
      hipFree(dev_out);
      return out;
}

bool check_error()
{
      hipError_t err = hipGetLastError();  
      return ( err != hipSuccess );
}

void copy_back(void* data, void*& d_data, int size)
{
      //hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
      hipFree(d_data);
      trensize-=size;
}

void copy_front(void* data, void*& d_data, int size)
{
      trensize+=size;
      hipMalloc(&d_data, size);
      //hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void g_alloc(void*& d_data, int size)
{
      hipMalloc(&d_data, size);
}

void just_return(void* data, void*& d_data, int size)
{
      hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
}

void just_front(void* data, void*& d_data, int size)
{
      hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
}

void dot_prodIntCPU(int* a, int* b, int* c, int n)
{
      dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
      int gridSize = (int)ceil((float)n/BLOCK_SIZE);
      dim3 Grid_dim(gridSize, gridSize);
      dotIntGPU << < Grid_dim, Block_dim >> > (a, b, c, n);
}
