#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "BasicOperations.h"

// Number of threads in each thread block
int blockSize = 1024;
 
__global__ void vecAddDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecMulDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecSubDoubleIntGPU(double *a, int *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntDoubleGPU(int *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}


__global__ void vecAddDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecAddIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] + b[id];
}

__global__ void vecSubDoubleGPU(double *a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecSubIntGPU(int *a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
 
      if (id < n)
            c[id] = a[id] - b[id];
}

__global__ void vecMulDoubleGPU(double* a, double *b, double *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

__global__ void vecMulIntGPU(int* a, int *b, int *c, int n)
{
      int id = blockIdx.x*blockDim.x+threadIdx.x;
      
      if (id < n)
            c[id] = a[id] * b[id];
}

void vecAddDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);

      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulDoubleCPU(double *a, double *b, double *c, int n)
{
      double *d_a;
      double *d_b;
      double *d_c;
      int bytes = sizeof(double)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecAddIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulIntCPU(int *a, int *b, int *c, int n)
{
      int *d_a;
      int *d_b;
      int *d_c;
      int bytes = sizeof(int)*n;
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
      hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecAddDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecAddDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecMulDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecMulDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubDoubleIntCPU(double *a, int *b, double *c, int n)
{
      double *d_a;
      int *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes2);
      hipMalloc(&d_b, bytes1);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes2, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes1, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubDoubleIntGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void vecSubIntDoubleCPU(int *a, double *b, double *c, int n)
{
      int *d_a;
      double *d_b;
      double *d_c;
      int bytes1 = sizeof(int)*n;
      int bytes2 = sizeof(double)*n;
      hipMalloc(&d_a, bytes1);
      hipMalloc(&d_b, bytes2);
      hipMalloc(&d_c, bytes2);
      hipMemcpy( d_a, a, bytes1, hipMemcpyHostToDevice);
      hipMemcpy( d_b, b, bytes2, hipMemcpyHostToDevice);
      int gridSize = (int)ceil((float)n/blockSize);
      vecSubIntDoubleGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
      hipMemcpy( c, d_c, bytes2, hipMemcpyDeviceToHost );
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
}

void initVecIntCPU(int *a, int x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}

extern void initVecDoubleCPU(double* a, double x, int n)
{
      for (int i=0;i<n;i++)
      {
            a[i]=x;
      }
}
